#include "hip/hip_runtime.h"
#pragma once
#ifdef __INTELLISENSE__
void __syncthreads();
#endif

#include "hip/hip_runtime.h"
#include ""
#include <thrust/device_vector.h>
#include <vector>
#include <iostream>
#include <chrono>

__global__ void print_arr(int* pd_vec, int n){
    if (threadIdx.x < n)
        printf("%d\n",pd_vec[threadIdx.x]);
}

__global__ void set_hist(int* hist) {
    hist[0] = 0;
    hist[1] = 0;
}

__global__ void radixSort_hist(int* d_input, int* d_output, int* hist, int size, int bit) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        int bitVal = (d_input[idx] >> bit) & 1;
        atomicAdd(&hist[bitVal], 1);
    }
    __syncthreads();
}

__global__ void radixSort_prefix_sum(int * hist) {
    int prefixsum = 0;
    for (int i = 0; i < 2; i += 1) {
        int temp = hist[i];
        hist[i] = prefixsum;
        prefixsum += temp;
    }
}

__global__ void radixSort_move(int* d_input, int* d_output, int* hist, int size, int bit) {
    for (int i = 0; i < size; i += 1) {
        int bitVal = (d_input[i] >> bit) & 1;
        int targetIdx = hist[bitVal]++;
        d_output[targetIdx] = d_input[i];
    }
}

extern "C" void Kernel(std::vector<int> &input_v) {
	int input_size = input_v.size();
	int block_size = 256;
	int num_block = input_size / block_size + 1;
 
	//cuda array pointer
    int* d_input = nullptr, * d_output = nullptr;

	//cuda memory allocation and copy memory to GPU
	hipMalloc((void**)&d_input, sizeof(int) * input_size);
	hipMemcpy(d_input, input_v.data(), input_size * sizeof(int), hipMemcpyHostToDevice);
    hipMalloc((void**)&d_output, sizeof(int) * input_size);

    //sort function call
    int* hist = nullptr;
    hipMalloc((void**)&hist, sizeof(int) * 3);

    std::chrono::high_resolution_clock::time_point start, end;
    start = std::chrono::high_resolution_clock::now();
    for (int bit = 0; bit < 32; ++bit) {
        //reset hist
        set_hist <<<1, 1>>> (hist);
        //create hist
        radixSort_hist<<<(input_size + block_size - 1) / block_size, block_size>>> (d_input, d_output, hist, input_size, bit);
        //calculate prefix sum
        radixSort_prefix_sum <<<1,1>>> (hist);
        //move element
        radixSort_move <<<1,1>>> (d_input, d_output, hist, input_size, bit);
        //radixSort_move << <(input_size + block_size - 1) / block_size, block_size >> > (d_input, d_output, hist, input_size, bit);
        std::swap(d_input, d_output); // Swap input and output arrays for next iteration
    }
    end = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
    std::cout << "execution time for GPU:" << duration.count() << std::endl;

    //copy data from GPU to original
    hipMemcpy(input_v.data(), d_input, input_size * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_input);
    hipFree(d_output);
    hipFree(hist);
}
